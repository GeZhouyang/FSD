#include "hip/hip_runtime.h"
// This file is part of the PSEv3 plugin, released under the BSD 3-Clause License
//
// Andrew Fiore

#include "Helper_Integrator.cuh"

#include "hoomd/Saru.h"
#include "hoomd/TextureTools.h"
using namespace hoomd;

#include <stdio.h>
#include <math.h>

#include "lapacke.h"
#include "cblas.h"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif


/*! 
	Helper_Integrator.cu

	Helper functions for saddle point integration
*/
	
/*!
  	Generate random numbers on particles.
	
	d_psi		(output) random vector
        n		(input)  number of particles
	timestep	(input)  length of time step
	seed		(input)  seed for random number generation

*/
__global__ void Integrator_RFD_RandDisp_kernel(
								float *d_psi,
								unsigned int N,
								const unsigned int seed
								){

	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	// Check if thread is in bounds
	if (idx < N) {

		// Initialize random seed
                detail::Saru s(idx, seed);

		// Square root of 3
		float sqrt3 = 1.732050807568877;
		
		// Call the random number generator
		float x1 = s.f( -sqrt3, sqrt3 );
		float y1 = s.f( -sqrt3, sqrt3 );
		float z1 = s.f( -sqrt3, sqrt3 );
		float x2 = s.f( -sqrt3, sqrt3 );
		float y2 = s.f( -sqrt3, sqrt3 );
		float z2 = s.f( -sqrt3, sqrt3 );

		// Write to output
		d_psi[ 6*idx + 0 ] = x1;
		d_psi[ 6*idx + 1 ] = y1;
		d_psi[ 6*idx + 2 ] = z1;
		d_psi[ 6*idx + 3 ] = x2;
		d_psi[ 6*idx + 4 ] = y2;
		d_psi[ 6*idx + 5 ] = z2; 

	}

}

/*! 
	The output velocity

	d_b	(output) output vector
   	N 	(input)  number of particles

*/
__global__ void Integrator_ZeroVelocity_kernel( 
						float *d_b,
						unsigned int N
						){

	// Thread index
	unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
	
	// Check if thread is inbounds
	if ( tid < N ) {
	
		d_b[ 6*tid + 0 ] = 0.0;
		d_b[ 6*tid + 1 ] = 0.0;
		d_b[ 6*tid + 2 ] = 0.0;
		d_b[ 6*tid + 3 ] = 0.0;
		d_b[ 6*tid + 4 ] = 0.0;
		d_b[ 6*tid + 5 ] = 0.0;
	
	}
}

/*! 
	Add rate of strain from shearing to the right-hand side of the saddle point solve

	d_b		(input/output) 	right-hand side vector
	shear_rate 	(input) 	shear rate of applied deformation
   	N 		(input)  	number of particles

*/
__global__ void Integrator_AddStrainRate_kernel( 
						float *d_b,
						float shear_rate,
						unsigned int N
						){

	// Thread index
	unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
	
	// Check if thread is inbounds
	if ( tid < N ) {


		// Index into array
		int ind = 6*N + 5*tid;

		// Add strain rate. For each particle, stores
		// [ F1, F2, F3, L1, L2, L3, E1, E2, E3, E4, E5 ]	
		d_b[ ind + 0 ] += 0.0;
		d_b[ ind + 1 ] += shear_rate; //zhoge: because it is 2E_xy, see "Computational tricks"
		d_b[ ind + 2 ] += 0.0;
		d_b[ ind + 3 ] += 0.0;
		d_b[ ind + 4 ] += 0.0;

	}
}
